#include "hip/hip_runtime.h"
#include "GPU/GPU_PBD_FEM_Coupled.h"
#include <GPU/CUMatrix.h>

__device__ void xpbd_solve_coupled(const int nb_vert_elem, const scalar stiffness1, const scalar stiffness2, const scalar dt, const scalar* C, const Vector3* grad_C, scalar* inv_mass, int* topology, Vector3* p, int* mask)
{
    const scalar a1 = 1.f / (stiffness1 * dt * dt);
    const scalar a2 = 1.f / (stiffness2 * dt * dt);
    Matrix2x2 A(a1,0,0,a2);
    for (int i = 0; i < nb_vert_elem; ++i) {
        const scalar wi = inv_mass[topology[i]];
        A[0][0] += glm::dot(grad_C[i], grad_C[i]) * wi;
        A[1][0] += glm::dot(grad_C[i+nb_vert_elem], grad_C[i]) * wi;
        A[1][1] += glm::dot(grad_C[i+nb_vert_elem], grad_C[i+nb_vert_elem]) * wi;
    }

    A[0][1] = A[1][0]; // 2x2 symmetric matrix
    Vector2 dt_lambda = -glm::inverse(A) * Vector2(C[0], C[1]);
    for (int i = 0; i < nb_vert_elem; ++i) {
        const int vid = topology[i];
        if(mask[vid] == 1)
            p[vid] += (dt_lambda[0] * grad_C[i] + dt_lambda[1] * grad_C[i+nb_vert_elem]) * inv_mass[vid];
    }
}


__device__ void xpbd_constraint_fem_eval_coupled(
    const int nb_vert_elem, const scalar lambda, const scalar mu, const Material material, const Matrix3x3& Jx_inv, const scalar& V,
    Vector3* dN, Vector3* p, int* topology, scalar* C, Vector3* grad_C)
{
    const Matrix3x3 Jx = compute_transform(nb_vert_elem, p, topology, dN);
    const Matrix3x3 F = Jx * Jx_inv;

    Matrix3x3 P; scalar energy;

    eval_material(material, 0, lambda, mu, F, P, energy);
    P = P * glm::transpose(Jx_inv) * V;
    C[0] += energy * V;
    for (int i = 0; i < nb_vert_elem; ++i) {
        grad_C[i] += P * dN[i];
    }

    eval_material(material, 1, lambda, mu, F, P, energy);
    P = P * glm::transpose(Jx_inv) * V;
    C[1] += energy * V;
    for (int i = 0; i < nb_vert_elem; ++i) {
        grad_C[nb_vert_elem + i] += P * dN[i];
    }
}


__global__ void kernel_XPBD_Coupled_V0(
    const int n, const int nb_quadrature, const int nb_vert_elem, const scalar dt, // some global data
    const scalar stiffness_1, const scalar stiffness_2, const Material material, // material
    const int offset, // coloration
    Vector3* cb_dN,
    Vector3 *cb_p, int *cb_topology, // mesh
    scalar *inv_mass, int* mask,
    scalar *cb_V, Matrix3x3 *cb_JX_inv // element data (Volume * Weight, Inverse of initial jacobian)
)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // thread it
    if (tid >= n) return;
    const int eid = tid + offset / nb_vert_elem;
    const int vid = tid * nb_vert_elem + offset; // first vertice id in topology
    const int qid = eid * nb_quadrature;
    int* topology = cb_topology+vid;


    Vector3 grad_C[64];
    scalar C[2] = {0.f, 0.f};

    for (int j = 0; j < nb_vert_elem * 2; ++j)
        grad_C[j] = Vector3(0, 0, 0);

    for (int q = 0; q < nb_quadrature; ++q) { // must be possible to do in parrallel
        Vector3* dN = cb_dN + q * nb_vert_elem;
        xpbd_constraint_fem_eval_coupled(nb_vert_elem, stiffness_1, stiffness_2, material, cb_JX_inv[qid + q], cb_V[qid + q], dN, cb_p, topology, C, grad_C);
    }
    xpbd_convert_to_constraint(nb_vert_elem, C[0], grad_C);
    xpbd_convert_to_constraint(nb_vert_elem, C[1], grad_C + nb_vert_elem);
    xpbd_solve_coupled(nb_vert_elem, stiffness_1, stiffness_2, dt, C, grad_C, inv_mass, topology, cb_p, mask);
}


__device__ void xpbd_constraint_fem_eval_coupled_V2(const Material material, const scalar lambda, const scalar mu, const Matrix3x3& Jx_inv, const Matrix3x3& F, const scalar& V, scalar* C, Matrix3x3* P)
{
    for(int m = 0; m < 2; ++m) {
        eval_material(material, m, lambda, mu, F, P[m], C[m]);
        P[m] = P[m] * glm::transpose(Jx_inv) * V;
        C[m] *= V;
    }
}

__global__ void kernel_XPBD_Coupled_V2(
    const int n, const int nb_quadrature, const int nb_vert_elem, const scalar dt, // some global data
    const scalar stiffness_1, const scalar stiffness_2, const Material material, // material
    const int offset, // coloration
    Vector3* cb_dN,
    Vector3 *cb_p, int *cb_topology, // mesh
    scalar *inv_mass,
    scalar *cb_V, Matrix3x3 *cb_JX_inv // element data (Volume * Weight, Inverse of initial jacobian)
)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // thread it
    if (tid >= n) return;
    const int eid = tid/nb_quadrature + offset / nb_vert_elem;
    const int vid = tid/nb_quadrature * nb_vert_elem + offset; // first vertice id in topology
    const int qid = eid * nb_quadrature;
    const int q = tid % nb_quadrature;
    int* topology = cb_topology+vid;

    __shared__ Matrix3x3 P[8];
    scalar C[8];

    for (int j = 0; j < nb_quadrature * 2; ++j) {
        P[j] = Matrix3x3(0.f);
        C[j] = 0.f;
    }

    Vector3* dN = cb_dN + q * nb_vert_elem;
    const Matrix3x3 F = compute_transform(nb_vert_elem, cb_p, topology, dN) * cb_JX_inv[qid + q];
    xpbd_constraint_fem_eval_coupled_V2(material, stiffness_1, stiffness_2, cb_JX_inv[qid + q], F, cb_V[qid + q], C+q*2, P+q*2);
    

}



void GPU_PBD_FEM_Coupled::step(const GPU_ParticleSystem* ps, const scalar dt) {
    for (int j = 0; j < c_offsets.size(); ++j) {
        kernel_XPBD_Coupled_V0<<<c_nb_elem[j],nb_quadrature>>>(c_nb_elem[j], nb_quadrature, elem_nb_vert, dt,
                                                       lambda, mu, _material,
                                                      c_offsets[j],
                                                      cb_dN->buffer,
                                                      ps->cb_position->buffer, cb_topology->buffer,
                                                      ps->cb_inv_mass->buffer, ps->cb_mask->buffer,
                                                      cb_V->buffer, cb_JX_inv->buffer);


    }
}
