#include "hip/hip_runtime.h"
#include "GPU/GPU_VBD.h"
#include <glm/detail/func_matrix_simd.inl>
#include <Manager/Debug.h>
#include <Manager/TimeManager.h>

__device__ Matrix3x3 vec_hat(const Vector3 &v) {
    return {
        0.f, -v.z, v.y,
        v.z, 0.f, -v.x,
        -v.y, v.x, 0.f
    };
}
__device__ void print_vec3(const Vector3 &v) {
    printf("(x:%f y:%f z:%f)", v.x, v.y, v.z);
}

__device__ void print_mat3(const Matrix3x3 &m) {
    printf("|%f %f %f|\n|%f %f %f|\n|%f %f %f|\n", m[0][0], m[1][0], m[2][0], m[0][1], m[1][1], m[2][1], m[0][2], m[1][2],
           m[2][2]);
}

__global__ void kernel_plane_fix(const int nb, scalar t, const Vector3 o, Vector3 n, Vector3 *p_init, Vector3 *y, Vector3 *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nb) return;

    Vector3 d = p_init[i] - o;
    if(glm::dot(d, n) > 0) {
        p[i] = p_init[i] + n * abs(cos(t+3.14f*0.5f)) * 0.f;
        y[i] = p[i];
    }
}

__global__ void kernel_integration(
        const int n, const scalar dt, const Vector3 g,
        Vector3 *p, Vector3 *prev_p, Vector3* y, Vector3* prev_it_p, Vector3 *v, Vector3 *f, scalar *w) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    prev_p[i] = p[i]; // x^t-1 = x^t
    prev_it_p[i] = p[i];
    const Vector3 a_ext = g + f[i] * w[i];
    y[i] = p[i] + (v[i] + a_ext * dt) * dt;
    p[i] = y[i];

    f[i] *= 0;
}

__global__ void kernel_velocity_update(int n, scalar dt, Vector3* prev_p, Vector3* p, Vector3* v, scalar* _inv_mass) {
    const int vid = blockIdx.x * blockDim.x + threadIdx.x;
    if(vid >= n) return;
    v[vid] = (p[vid] - prev_p[vid]) / dt;
    //if(vid == 10) printf("v(%f %f %f)\n", v[vid].x, v[vid].y, v[vid].z);
}

__global__ void kernel_chebychev_acceleration(int n, int it, scalar omega, Vector3* prev_it_p, Vector3* prev_it2_p, Vector3* p) {
    const int vid = blockIdx.x * blockDim.x + threadIdx.x;
    if(vid >= n) return;
    if(it >= 2) {
        p[vid] = prev_it2_p[vid] + omega * (p[vid] - prev_it2_p[vid]);
    }
    prev_it2_p[vid] = prev_it_p[vid];
    prev_it_p[vid] = p[vid];
}

__global__ void kernel_solve(
    // nb_thread, nb quadrature per elements, nb vertices in element
    const int n, const int nb_quadrature, const int elem_nb_verts,
    const scalar lambda, const scalar mu, const scalar damping, const scalar dt,
    int offset,
    int *nb_owners, // nb_vertices
    int *owner_off, // nb_vertices
    int *owners, // nb_neighbors.size()
    int *ref_vid, // nb_neighbors.size()
    int *topology, // nb_element * elem_nb_vert
    Vector3 *y, // nb_vertices
    Vector3 *p, // nb_vertices
    Vector3 *prev_p, // nb_vertices
    Vector3 *f,
    scalar *mass, // nb_vertices
    Vector3 *dN, // elem_nb_verts * nb_quadrature
    Matrix3x3 *JX_inv, // nb_element * nb_quadrature
    scalar *V // nb_element * nb_quadrature
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains this vertices
    // and the number of needed quadratures
    const int cid = offset + blockIdx.x; // vertex position in coloration
    const int size_of_block = nb_owners[cid] * nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % nb_quadrature; // quadrature number
    const int e_off = owner_off[cid] + tid / nb_quadrature; // offset in buffer to find the right element
    const int eid = owners[e_off]; // element id
    const int r_vid = ref_vid[e_off]; // vertex id in this element

    const int *topo = topology + eid * elem_nb_verts; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    const int qe_off = eid * nb_quadrature + qid;
    const int qv_off = qid * elem_nb_verts;
    //if(threadIdx.x == 0) printf("[%d][%d/%d] cid=%d, nb=%d, offset=%d, vid=%d, eid=%d, qid=%d, rid=%d, qe_off=%d, qv_off=%d \n",gid,tid+1,size_of_block,cid, nb_owners[cid],offset,vid, eid, qid, r_vid, qe_off, qv_off);
    Matrix3x3 Jx(0.f);
    Matrix3x3 d2W_dF2[9];

    for (int i = 0; i < elem_nb_verts; ++i) {
        Jx += glm::outerProduct(p[topo[i]], dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * JX_inv[qe_off];


    /*
    //Hooke
    //force
    const Matrix3x3 e = 0.5f * (glm::transpose(F) + F ) - Matrix3x3(1.f);
    const Matrix3x3 P = lambda * (e[0][0]+e[1][1]+e[2][2]) * Matrix3x3(1.f) + mu * e;

    //Hessian
    for(int i = 0; i < 9; ++i) { d2W_dF2[i] = Matrix3x3(0); }
    for(int i = 0; i < 3; ++i) {
        d2W_dF2[i * 4] = Matrix3x3(lambda + mu);
    }
    */

    // Neohooke
    // Force
    const scalar detF = glm::determinant(F);
    const scalar alpha = 1.f + mu / lambda;
    Matrix3x3 comF(0);
    comF[0] = glm::cross(F[1], F[2]);
    comF[1] = glm::cross(F[2], F[0]);
    comF[2] = glm::cross(F[0], F[1]);
    const Matrix3x3 P = mu * F + lambda * (detF - alpha) * comF;
    // H = sum mi / h^2 I + sum d^2W / dxi^2
    scalar s = lambda * (detF - alpha);
    // lambda * (I3 - alpha) * H3
    d2W_dF2[0] = Matrix3x3(0);
    d2W_dF2[1] = vec_hat(F[2]) * s;
    d2W_dF2[2] = -vec_hat(F[1]) * s;
    d2W_dF2[3] = -d2W_dF2[1];
    d2W_dF2[4] = Matrix3x3(0);
    d2W_dF2[5] = vec_hat(F[0]) * s;
    d2W_dF2[6] = -d2W_dF2[2];
    d2W_dF2[7] = -d2W_dF2[5];
    d2W_dF2[8] = Matrix3x3(0);

    // mu/2 * H2 = mu * I_9x9x
    for (int i = 0; i < 3; ++i) {
        d2W_dF2[0][i][i] += mu;
        d2W_dF2[4][i][i] += mu;
        d2W_dF2[8][i][i] += mu;
    }

    // lambda vec(com F) * vec(com F)^T
    for (int i = 0; i < 3; ++i)
        for (int j = 0; j < 3; ++j)
            d2W_dF2[i*3 + j] += glm::outerProduct(comF[i], comF[j]) * lambda;

    const Vector3 dF_dx = glm::transpose(JX_inv[qe_off]) * dN[qv_off + r_vid];
    // Compute force at vertex i
    Vector3 fi = -P * dF_dx * V[qe_off];

    // assemble hessian
    Matrix3x3 H;
    for (int j = 0; j < 3; ++j) {
        for (int i = 0; i < 3; ++i) {
            Matrix3x3 H_kl;
            for(int l = 0; l < 3; ++l) {
                for(int k = 0; k < 3; ++k) {
                    H_kl[k][l] = d2W_dF2[k+l*3][i][j];
                }
            }
            H[i][j] = glm::dot(dF_dx, H_kl * dF_dx) * V[qe_off];
        }
    }

    // shared variable : f, H
    // we can do a much better reduction (without atomic add with a shared buffer)

    __shared__ __builtin_align__(16) scalar s_f_H[1024]; // size = block_size * 12 * sizeof(float)
    for(int i = 0; i < 3; ++i) {
        s_f_H[tid * 12 + i] = fi[i];
        //s_f_H[tid * 12 + i] = 1;
        for(int j = 0; j < 3; ++j) {
            s_f_H[tid * 12 + (i+1)*3 + j] = H[i][j];
            //s_f_H[tid * 12 + (i+1)*3 + j] = 1;
        }
    }
    //printf("%d < %d\n", tid, size_of_block);

    __syncthreads();
    int t = size_of_block;
    int i,b;
    for(i=t/2, b=(t+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            for(int j = 0; j < 12; ++j) {
                s_f_H[tid*12+j] += s_f_H[(tid+b)*12+j];
            }
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }

    if (threadIdx.x == 0) {
        H[0][0] = s_f_H[3]; H[0][1] = s_f_H[6]; H[0][2] = s_f_H[9];
        H[1][0] = s_f_H[4]; H[1][1] = s_f_H[7]; H[1][2] = s_f_H[10];
        H[2][0] = s_f_H[5]; H[2][1] = s_f_H[8]; H[2][2] = s_f_H[11];
        fi.x = s_f_H[0]; fi.y = s_f_H[1]; fi.z = s_f_H[2];

        // damping
        fi += -(damping / dt) * H * (p[vid] - prev_p[vid]);
        H += damping / dt * H;

        // intertia
        scalar mh2 = mass[vid] / (dt*dt);
        fi += -mh2 * (p[vid] - y[vid]);
        H[0][0] += mh2;
        H[1][1] += mh2;
        H[2][2] += mh2;

        //scalar detH = glm::determinant(s_H);
        //Vector3 dx = detH > 1e-6f ? glm::inverse(s_H) * s_f : Vector3(0.f);

        scalar detH = glm::determinant(H);
        Vector3 dx = detH > 1e-6f ? glm::inverse(H) * fi : Vector3(0.f);
        p[vid] += dx;
    }
}



void GPU_VBD_FEM::step(const GPU_VBD* vbd, const scalar dt, const scalar damping) {
    for(int c = 0; c < nb_color; ++c) {
        int grid_size = (c_nb_threads[c]+c_block_size[c]-1)/c_block_size[c];

        kernel_solve<<<grid_size, c_block_size[c]>>>(
        c_nb_threads[c], nb_quadrature, elem_nb_vert, lambda, mu, damping, dt, c_offsets[c],
        cb_nb_neighbors->buffer, cb_neighbors_offset->buffer, cb_neighbors->buffer, cb_ref_vid->buffer,
        cb_topology->buffer,
        vbd->y->buffer, vbd->cb_position->buffer,vbd->cb_prev_position->buffer, vbd->cb_forces->buffer, vbd->cb_mass->buffer,
        cb_dN->buffer, cb_JX_inv->buffer, cb_V->buffer
        );
    }
}

void GPU_VBD::step(const scalar dt) const {
    Time::Tic();
    const scalar r = 0.8;
    const scalar sub_dt = dt / static_cast<scalar>(sub_iteration);
    Vector3 v = Unit3D::right();
    for(int i = 0; i < sub_iteration; ++i) {
        scalar omega = 1;
        // integration / first guess
        kernel_integration<<<(n + 255)/256, 256>>>(n,sub_dt,Dynamic::gravity(),
            cb_position->buffer,cb_prev_position->buffer,y->buffer, prev_it_p->buffer,
            cb_velocity->buffer,cb_forces->buffer, cb_inv_mass->buffer);

        for(int j = 0; j < iteration; ++j) {
            // solve
            dynamic->step(this, sub_dt, _damping);
            kernel_plane_fix<<<(n + 255)/256, 256>>>(n, Time::Fixed_Timer(), v*0.01f, -v, cb_init_position->buffer, y->buffer, cb_position->buffer);
            //kernel_plane_fix<<<(n + 255)/256, 256>>>(n, Time::Fixed_Timer(), v*1.99f, v, cb_init_position->buffer, y->buffer, cb_position->buffer);
            // Acceleration (Chebychev)
            if(j == 1) omega = 2.f / (2.f - r * r);
            else if(j > 1) omega = 4.f / (4.f - r * r * omega);
            //kernel_chebychev_acceleration<<<(n + 255)/256, 256>>>(n, j, omega, prev_it_p->buffer, prev_it2_p->buffer, cb_position->buffer);
        }
        // velocity update
        kernel_velocity_update<<<(n + 255)/256, 256>>>(n,sub_dt,
            cb_prev_position->buffer, cb_position->buffer, cb_velocity->buffer, cb_inv_mass->buffer);
    }
    hipDeviceSynchronize();
    scalar time = Time::Tac() *1000.f;
    DebugUI::Begin("VBD");
    DebugUI::Plot("Time vbd", time);
    DebugUI::Value(" ", time);
    DebugUI::Range("", time);
    DebugUI::End();
}

GPU_VBD::~GPU_VBD() {
    delete integrator;
    delete dynamic;
}