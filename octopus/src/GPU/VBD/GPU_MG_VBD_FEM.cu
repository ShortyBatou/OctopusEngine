#include "hip/hip_runtime.h"
#include "GPU/VBD/GPU_MG_VBD_FEM.h"
#include <Dynamic/VBD/MG_VertexBlockDescent.h>
#include <glm/detail/func_matrix_simd.inl>
#include <Manager/Debug.h>
#include <random>
#include <numeric>
#include <Dynamic/FEM/FEM_Generic.h>
#include <GPU/CUMatrix.h>


__global__ void kernel_prolongation(const int n, GPU_ParticleSystem_Parameters ps, GPU_MG_Interpolation_Parameters inter) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    const int* primitive = inter.primitives + tid * inter.nb_vert_primitives;
    Vector3 dt_p = Vector3(0);
    for(int i = 0; i < inter.nb_vert_primitives; ++i)
    {
        const int vid = primitive[i];
        dt_p += ps.p[vid] - ps.last_p[vid];
    }
    ps.p[inter.ids[tid]] = ps.last_p[inter.ids[tid]] + dt_p * inter.weight;
}

__global__ void kernel_restriction_intertia(
    const int n, const scalar dt, const Vector3 g,
    GPU_ParticleSystem_Parameters ps, GPU_MG_Interpolation_Parameters inter,
    Vector3* y) {

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    const int* primitive = inter.primitives + tid * inter.nb_vert_primitives;
    Vector3 v = Vector3(0);
    const int vid = inter.ids[tid];
    for(int i = 0; i < inter.nb_vert_primitives; ++i)
    {
        const int p_vid = primitive[i];
        v += ps.v[p_vid] * ps.m[p_vid] / ps.m[vid];
    }
    v *= 1.f / inter.nb_vert_primitives;

    const Vector3 a_ext = g + ps.f[vid] * ps.w[vid];
    y[vid] = ps.last_p[vid] + (ps.v[vid] + a_ext * dt) * dt;
}

__global__ void kernel_intertia(
        const scalar dt, const Vector3 g,
        GPU_ParticleSystem_Parameters ps,
        Vector3* y) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ps.nb_particles) return;
    const Vector3 a_ext = g + ps.f[i] * ps.w[i];
    y[i] = ps.last_p[i] + (ps.v[i] + a_ext * dt) * dt;
}


GPU_MG_VBD_FEM::GPU_MG_VBD_FEM(const Element& element, const Mesh::Topology& topology, const Mesh::Geometry& geometry,
                               const Material& material, const scalar& young, const scalar& poisson,
                               const scalar& damping, const scalar& linear, const int& nb_iteration,
                               const scalar& density, const Mass_Distribution& mass_distrib,
                               GPU_ParticleSystem* ps) :
    GPU_VBD_FEM(element, topology, geometry, material, young, poisson, damping)
{
    assert(element == Tetra10 || element == Hexa27);

    int it_linear = static_cast<int>(static_cast<scalar>(nb_iteration) * linear);
    int it_quad = nb_iteration - it_linear;
    nb_iterations = std::vector<int>({it_quad, it_linear});
    it_count = 0;
    level = 1;


    interias.push_back(new Cuda_Buffer(geometry));
    interias.push_back(new Cuda_Buffer(geometry));
    y = interias[level];
    const int nb_vert_elem = elem_nb_vertices(element);
    const int nb_elem = static_cast<int>(topology.size()) / nb_vert_elem;

    // get linear topology (could be nice to have that in a global function)
    const Element lin_elem = get_linear_element(element);
    const int lin_nb_vert_elem = elem_nb_vertices(lin_elem);
    std::set<int> vids;
    std::vector<int> lin_topo(nb_elem * lin_nb_vert_elem);
    for (int i = 0; i < nb_elem; i++)
    {
        for (int j = 0; j < lin_nb_vert_elem; ++j)
        {
            int vid = topology[i * nb_vert_elem + j];
            lin_topo[i * lin_nb_vert_elem + j] = vid;
            vids.insert(vid);
        }
    }

    // fem data of quadratic and linear element
    l_fems.push_back(d_fem);
    l_fems.push_back(GPU_FEM::build_fem_const(lin_elem, geometry, lin_topo));

    // thread data
    l_threads.push_back(d_thread);
    l_threads.push_back(new Thread_Data());

    l_owners.push_back(d_owners);
    l_owners.push_back(new GPU_Owners_Data());
    // prepare FEM to build new thread data for linear fem
    d_thread = l_threads.back();
    d_fem = l_fems.back();
    d_owners = l_owners.back();

    // create linear data (modify d_thread, d_owners and d_block <== this one is useless for now)
    std::vector<std::vector<int>> e_owners;
    std::vector<std::vector<int>> e_ref_id;
    build_owner_data(vids.size(), lin_topo, e_owners, e_ref_id);
    Coloration coloration = build_graph_color(lin_elem, lin_topo); // get coloration
    create_buffers(lin_elem, lin_topo, coloration, e_owners, e_ref_id);

    masses.push_back(ps->_data->_cb_mass);
    std::vector<scalar> lin_masses = compute_fem_mass(lin_elem, geometry, lin_topo, density, mass_distrib);
    masses.push_back(new Cuda_Buffer<scalar>(lin_masses));

    // build interpolations
    if(element == Tetra10)
    {
        P1_to_P2 inter(topology);
        auto* i_mid_edge = new GPU_MG_Interpolation(2,0.5, inter.ids, inter.edges);
        interpolations.push_back(i_mid_edge);
    }
    else
    {
        Q1_to_Q2 inter(topology);
        auto* i_mid_edge = new GPU_MG_Interpolation(2,0.5, inter.ids_edges, inter.edges);
        interpolations.push_back(i_mid_edge);

        auto* i_mid_face = new GPU_MG_Interpolation(4,0.25,inter.ids_faces, inter.faces);
        interpolations.push_back(i_mid_face);

        auto* i_mid_volume = new GPU_MG_Interpolation(8,0.125,inter.ids_volumes, inter.volume);
        interpolations.push_back(i_mid_volume);
    }

    FEM_Shape* q_shape = get_fem_shape(Tetra20);
    FEM_Shape* shape = get_fem_shape(element);
    FEM_Shape* lin_shape = get_fem_shape(lin_elem);
    std::vector<scalar> masses = compute_fem_mass(element, geometry, topology, density, mass_distrib);
    std::vector<scalar> quad_coord = q_shape->get_quadrature_coordinates();
    std::vector<Vector3> verts(shape->nb);

    // sparse matrix
    std::map<std::pair<int, int>, scalar> proj;

    // compute the projection matrix with Gauss quadratures
    for(int q = 0; q < q_shape->nb_quadratures(); ++q) {
        scalar w = q_shape->weights[q];
        scalar x = quad_coord[q * 3], y = quad_coord[q * 3 + 1], z = quad_coord[q * 3 + 2];
        std::vector<scalar> shape_func = shape->build_shape(x,y,z);
        std::vector<scalar> lin_shape_func = lin_shape->build_shape(x,y,z);
        std::vector<Vector3> dN = shape->build_shape_derivatives(x, y, z);

        // get the projection for each element
        for (int i = 0; i < topology.size(); i += nb_vert_elem)
        {
            std::vector<int> e_topo(topology.begin() + i, topology.begin() + i + nb_vert_elem);
            for(int j = 0; j < shape->nb; j++) verts[j] = geometry[e_topo[j]];
            const scalar v = glm::determinant(FEM_Generic::get_jacobian(verts, dN));
            const scalar volume = abs(v) * w;

            // compute each value in projection matrix
            for(int k = 0; k < shape->nb; ++k) {
                int vid = e_topo[k];
                for(int l = 0; l < lin_shape->nb; ++l) {
                    std::pair<int, int> pair(vid, e_topo[l]);
                    scalar p = shape_func[k] * lin_shape_func[l] * volume  / (masses[vid] / density);
                    if(proj.find(pair) != proj.end()) proj[pair] += p;
                    else proj[pair] = p;
                }
            }
        }
    }

    delete shape;
    delete lin_shape;
}


void GPU_MG_VBD_FEM::compute_intertia(GPU_ParticleSystem* ps, const scalar dt) const {
    kernel_intertia<<<(ps->nb_particles() + 31)/32, 32>>>(dt,Dynamic::gravity(),
        ps->get_parameters(), y->buffer);
}

void GPU_MG_VBD_FEM::step(GPU_ParticleSystem* ps, const scalar dt)
{
    const auto ps_param = ps->get_parameters();

    it_count++;
    const int last_level = level;
    while(it_count > nb_iterations[level])
    {
        it_count = 1;
        level = (level + 1) % static_cast<int>(nb_iterations.size());
    }

    if (last_level != level && level == 1)
    {
        for(int i = 0; i < interpolations.size(); ++i)
        {
            const auto inter_param = get_interpolation_parameters(i);
            kernel_prolongation<<<(inter_param.nb_ids+31)/32,32>>>(inter_param.nb_ids, ps_param, inter_param);
        }
    }

    ps->_data->_cb_mass = masses[level];
    d_thread = l_threads[level];
    d_fem = l_fems[level];
    d_owners = l_owners[level];
    y = interias[level];

    GPU_VBD_FEM::step(ps, dt);

    if (level == 1)
    {
        for(int i = 0; i < interpolations.size(); ++i)
        {
            const auto inter_param = get_interpolation_parameters(i);
            kernel_prolongation<<<(inter_param.nb_ids+31)/32,32>>>(inter_param.nb_ids, ps_param, inter_param);
        }
    }
}
