#include "hip/hip_runtime.h"
#include "GPU/VBD/GPU_Mixed_VBD_FEM.h"

#include <GPU/CUMatrix.h>
#include <GPU/GPU_FEM_Material.h>
#include <GPU/Explicit/GPU_Explicit.h>
#include <Manager/TimeManager.h>

__global__ void kernel_explicit_fem_eval_force_2(
    // nb_thread, nb quadrature per elements, nb vertices in element
    const int n, const scalar damping,
    const Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners,
    scalar* w_max
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains this vertices
    // and the number of needed quadratures
    const int cid = blockIdx.x; // vertex position
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Matrix3x3 Jx(0.f);

    for (int i = 0; i < fem.elem_nb_vert; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * fem.JX_inv[qe_off];

    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F);

    const Vector3 dF_dx = glm::transpose(fem.JX_inv[qe_off]) * fem.dN[qv_off + r_vid];
    // Compute force at vertex i
    Vector3 fi = -P * dF_dx * fem.V[qe_off];

    // assemble hessian
    Matrix3x3 d2W_dF2[6];
    eval_hessian(mt.material, mt.lambda, mt.mu, F, d2W_dF2);
    Matrix3x3 H = assemble_sub_hessian(dF_dx, fem.V[qe_off], d2W_dF2);

    //Matrix3x3 H2 = glm::outerProduct(fi, fi);
    fi -= damping * H * ps.v[vid];/**/

    scalar l_w_max = 0;
    for(int i = 0; i < 3; ++i)
    {
        scalar w = 0;
        for(int j = 0; j <3; ++j)
        {
            w+= fabsf(H[i][j]);
        }
        l_w_max = fmaxf(l_w_max, w);
    }

    // shared variable : f, H
    __shared__ Vector3 s_f_H[256]; // size = block_size * 3 * sizeof(float)
    __shared__ scalar s_w_max[64]; // size = block_size * sizeof(float)
    s_f_H[tid] = fi;
    s_w_max[tid] = l_w_max;

    __syncthreads();
    const int t = size_of_block;
    int i,b;
    for(i=t/2, b=(t+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            s_f_H[tid] += s_f_H[tid+b];
            s_w_max[tid] += s_w_max[tid+b] ;
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }

    if (threadIdx.x == 0) {
        ps.f[vid] = s_f_H[0];
        w_max[vid] = s_w_max[0] * ps.w[vid];
    }
}

__global__ void kernel_explicit_fem_eval_force_3(
    // nb_thread, nb quadrature per elements, nb vertices in element
    const int n,
    const Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains this vertices
    // and the number of needed quadratures
    const int cid = blockIdx.x; // vertex position
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Matrix3x3 Jx(0.f);
    for (int i = 0; i < fem.elem_nb_vert; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * fem.JX_inv[qe_off];
    const Vector3 dF_dx = glm::transpose(fem.JX_inv[qe_off]) * fem.dN[qv_off + r_vid];
    // Compute force at vertex i
    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F);
    const Vector3 fi = -P * dF_dx * fem.V[qe_off];

    // shared variable : f, H
    __shared__ Vector3 s_f_H[256]; // size = block_size * 3 * sizeof(float)
    s_f_H[tid] = fi;

    __syncthreads();
    const int t = size_of_block;
    int i,b;
    for(i=t/2, b=(t+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            s_f_H[tid] += s_f_H[tid+b];
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }

    if (threadIdx.x == 0) {
        ps.f[vid] = s_f_H[0];
    }
}


__global__ void kernel_explicit_fem_eval_force_4(const int n, Material_Data mt, GPU_ParticleSystem_Parameters ps, GPU_FEM_Pameters fem, Vector3* p_forces)
{
    const int eid = blockIdx.x * blockDim.x + threadIdx.x;

    if (eid >= n) return;
    const int e_off = eid * fem.elem_nb_vert;
    const int qe_off = eid * fem.nb_quadrature;
    const int* topo = fem.topology + e_off;

    Vector3 s_f[27];
    for(int i = 0; i < fem.elem_nb_vert; ++i) s_f[i] = Vector3(0);
    for(int q = 0; q < fem.nb_quadrature; ++q)
    {
        Matrix3x3 Jx(0.f);
        for (int i = 0; i < fem.elem_nb_vert; ++i) {
            Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[q * fem.elem_nb_vert + i]);
        }
        const Matrix3x3 F = Jx * fem.JX_inv[qe_off + q];
        // Compute force at vertex i
        const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F) * glm::transpose(fem.JX_inv[qe_off + q]) * fem.V[qe_off + q];

        for(int i = 0; i < fem.elem_nb_vert; ++i)
        {
            s_f[i] -= P * fem.dN[q * fem.elem_nb_vert + i];
        }
    }
    for(int i = 0; i < fem.elem_nb_vert; ++i)
    {
        p_forces[e_off+i] = s_f[i];
    }
}

__global__ void kernel_explicit_fem_eval_force_5(const int n, Material_Data mt, GPU_ParticleSystem_Parameters ps, GPU_FEM_Pameters fem, Vector3* p_forces)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid >= n) return;
    const int block_size = fem.nb_quadrature;
    if(threadIdx.x >= block_size) return;
    const int q = threadIdx.x;
    const int eid = gid / block_size;
    const int e_off = eid * fem.elem_nb_vert;
    const int qe_off = eid * fem.nb_quadrature;
    const int qv_off = q * fem.elem_nb_vert;
    const int* topo = fem.topology + e_off;

    __shared__ Vector3 s_f[729];
    for(int i = 0; i < fem.elem_nb_vert; ++i) s_f[i] = Vector3(0);

    Matrix3x3 Jx(0.f);
    for (int i = 0; i < fem.elem_nb_vert; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * fem.JX_inv[qe_off + q];
    // Compute force at vertex i
    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F) * glm::transpose(fem.JX_inv[qe_off + q]) * fem.V[qe_off + q];

    for(int i = 0; i < fem.elem_nb_vert; ++i)
        s_f[qv_off+i] = -P * fem.dN[qv_off+ i];

    __syncthreads();
    for(int i = 0; i < fem.elem_nb_vert; i+=block_size)
    {
        const int r_vid = i + q;
        if(r_vid < fem.elem_nb_vert)
        {
            for(int j = 1; j < fem.nb_quadrature; j++)
            {
                s_f[r_vid] += s_f[fem.elem_nb_vert*j + r_vid];
            }
        }
        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        for(int i = 0; i < fem.elem_nb_vert; ++i)
        {
            p_forces[e_off+i] = s_f[i];
        }
    }
}

__global__ void kernel_explicit_fem_sum_partial_forces(const int n, GPU_ParticleSystem_Parameters ps, GPU_FEM_Pameters fem, GPU_Owners_Parameters owners, const Vector3* p_forces)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;

    Vector3 fi = Vector3(0);
    const int e_off = owners.offset[tid];
    const int nb_owner = owners.nb[tid];
    int eid = 0; // element id
    int r_vid = 0; // vertex id in this element
    for(int i = 0; i < nb_owner; ++i)
    {
        eid = owners.eids[e_off + i]; // element id
        r_vid = owners.ref_vid[e_off + i]; // vertex id in this element
        fi += p_forces[eid * fem.elem_nb_vert + r_vid];
    }

    const int vid = fem.topology[eid * fem.elem_nb_vert + r_vid];
    ps.f[vid] = fi;
}

GPU_Mixed_VBD_FEM::GPU_Mixed_VBD_FEM(const Element &element, const Mesh::Topology &topology, const Mesh::Geometry &geometry,
                         const Material& material, const scalar &young, const scalar &poisson, const scalar& damping) :
    GPU_VBD_FEM(element, topology, geometry, material, young, poisson, damping)
{

    p_forces = new Cuda_Buffer<Vector3>(std::vector<Vector3>(topology.size()));
    d_exp_thread = new Thread_Data();
    int block_size = 0;
    for(int i = 0; i < d_thread->nb_kernel; ++i)
    {
        block_size = std::max(block_size, d_thread->block_size[i]);
    }
    d_exp_thread->nb_kernel = 1;
    d_exp_thread->block_size.push_back(block_size);
    d_exp_thread->nb_threads.push_back(static_cast<int>(geometry.size()) * block_size);
    d_exp_thread->grid_size.push_back((d_exp_thread->nb_threads[0] + block_size-1) / block_size);
    d_exp_thread->offsets.push_back(0);
}

void GPU_Mixed_VBD_FEM::explicit_step(const GPU_ParticleSystem* ps, Cuda_Buffer<scalar>* w_max, scalar dt) const
{
    if(d_fem->elem_nb_vert == 4) {
        // if we use w_max has filter
        /*kernel_explicit_fem_eval_force_2<<<d_exp_thread->grid_size[0], d_exp_thread->block_size[0]>>>(
            d_exp_thread->nb_threads[0], _damping, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters(), w_max->buffer
        );*/

        kernel_explicit_fem_eval_force_3<<<d_exp_thread->grid_size[0], d_exp_thread->block_size[0]>>>(
            d_exp_thread->nb_threads[0], *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters()
        );/**/
    }/**/
    else {
        int block_size = 32;
        int nb_thread = d_fem->nb_element;
        int grid_size = (nb_thread + block_size-1) / block_size;
        kernel_explicit_fem_eval_force_4<<<grid_size, block_size>>>(nb_thread, *d_material, ps->get_parameters(), get_fem_parameters(), p_forces->buffer);/**/

        /*int block_size = d_fem->nb_quadrature;
          int nb_thread = d_fem->nb_element * d_fem->nb_quadrature;
          int grid_size = (nb_thread + block_size-1) / block_size;
          kernel_explicit_fem_eval_force_5<<<grid_size, block_size>>>(nb_thread, *d_material, ps->get_parameters(), get_fem_parameters(), p_forces->buffer);/**/

        block_size = 32;
        nb_thread = ps->nb_particles(); // nb_vertices
        grid_size = (nb_thread+block_size-1)/block_size;
        kernel_explicit_fem_sum_partial_forces<<<grid_size, block_size>>>(nb_thread, ps->get_parameters(), get_fem_parameters(), get_owners_parameters(), p_forces->buffer);/**/
    }
}
