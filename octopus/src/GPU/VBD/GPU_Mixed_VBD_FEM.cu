#include "hip/hip_runtime.h"
#include "GPU/VBD/GPU_Mixed_VBD_FEM.h"

#include <GPU/GPU_FEM_Material.h>
#include <GPU/Explicit/GPU_Explicit.h>


__global__ void kernel_explicit_fem_eval_force_2(
    // nb_thread, nb quadrature per elements, nb vertices in element
    const int n, const scalar damping,
    const Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners,
    scalar* w_max
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains this vertices
    // and the number of needed quadratures
    const int cid = blockIdx.x; // vertex position
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Matrix3x3 Jx(0.f);

    for (int i = 0; i < fem.elem_nb_vert; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * fem.JX_inv[qe_off];

    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F);

    const Vector3 dF_dx = glm::transpose(fem.JX_inv[qe_off]) * fem.dN[qv_off + r_vid];
    // Compute force at vertex i
    Vector3 fi = -P * dF_dx * fem.V[qe_off];

    // assemble hessian
    Matrix3x3 d2W_dF2[6];
    eval_hessian(mt.material, mt.lambda, mt.mu, F, d2W_dF2);
    Matrix3x3 H = assemble_sub_hessian(dF_dx, fem.V[qe_off], d2W_dF2);

    //Matrix3x3 H2 = glm::outerProduct(fi, fi);
    fi -= damping * H * ps.v[vid];/**/

    scalar l_w_max = 0;
    for(int i = 0; i < 3; ++i)
    {
        scalar w = 0;
        for(int j = 0; j <3; ++j)
        {
            w+= fabsf(H[i][j]);
        }
        l_w_max = fmaxf(l_w_max, w);
    }

    // shared variable : f, H
    __shared__ Vector3 s_f_H[256]; // size = block_size * 3 * sizeof(float)
    __shared__ scalar s_w_max[64]; // size = block_size * sizeof(float)
    s_f_H[tid] = fi;
    s_w_max[tid] = l_w_max;

    __syncthreads();
    const int t = size_of_block;
    int i,b;
    for(i=t/2, b=(t+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            s_f_H[tid] += s_f_H[tid+b];
            s_w_max[tid] += s_w_max[tid+b] ;
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }

    if (threadIdx.x == 0) {
        ps.f[vid] = s_f_H[0];
        w_max[vid] = s_w_max[0] * ps.w[vid];
    }
}

__global__ void kernel_explicit_fem_eval_force_3(
    // nb_thread, nb quadrature per elements, nb vertices in element
    const int n,
    const Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains this vertices
    // and the number of needed quadratures
    const int cid = blockIdx.x; // vertex position
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Matrix3x3 Jx(0.f);
    for (int i = 0; i < fem.elem_nb_vert; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], fem.dN[qv_off + i]);
    }
    const Matrix3x3 F = Jx * fem.JX_inv[qe_off];
    const Vector3 dF_dx = glm::transpose(fem.JX_inv[qe_off]) * fem.dN[qv_off + r_vid];
    // Compute force at vertex i
    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F);
    const Vector3 fi = -P * dF_dx * fem.V[qe_off];

    // shared variable : f, H
    __shared__ Vector3 s_f_H[256]; // size = block_size * 3 * sizeof(float)
    s_f_H[tid] = fi;

    __syncthreads();
    const int t = size_of_block;
    int i,b;
    for(i=t/2, b=(t+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            s_f_H[tid] += s_f_H[tid+b];
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }

    if (threadIdx.x == 0) {
        ps.f[vid] = s_f_H[0];
    }
}

void GPU_Mixed_VBD_FEM::explicit_step(GPU_ParticleSystem* ps, Cuda_Buffer<scalar>* w_max, scalar dt) const
{
     /*kernel_explicit_fem_eval_force_2<<<d_exp_thread->grid_size[0], d_exp_thread->block_size[0]>>>(
        d_exp_thread->nb_threads[0], _damping, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters(), w_max->buffer
    );/**/


    kernel_explicit_fem_eval_force_3<<<d_exp_thread->grid_size[0], d_exp_thread->block_size[0]>>>(
        d_exp_thread->nb_threads[0], *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters()
    );/**/
}
