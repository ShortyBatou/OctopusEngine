#include "hip/hip_runtime.h"
#include "GPU/VBD/GPU_Mixed_VBD.h"
#include <glm/detail/func_matrix_simd.inl>
#include <GPU/CUMatrix.h>
#include <Manager/Debug.h>
#include <Manager/Dynamic.h>


__global__ void kernel_rk4(
    const int n, const scalar dt, const Vector3 g, const int step,
    GPU_ParticleSystem_Parameters ps, Vector3* l, Vector3* k, Vector3* x_t)
{
    const int vid = blockIdx.x * blockDim.x + threadIdx.x;
    if (vid >= n || ps.mask[vid] == 0) return;
    const int i = vid * 4 + step;
    if(step == 0) {
        x_t[vid] = ps.p[vid];
        k[i] = ps.v[vid] * dt; // dt position
        l[i] = (ps.f[vid] * ps.w[vid] + g) * dt; //dt velocity

        ps.p[vid] += k[i] * 0.5f;
    }
    else if(step == 1 || step == 2)
    {
        k[i] = (ps.v[vid] + 0.5f * l[i-1]) * dt; //dt position
        l[i] = (ps.f[vid] * ps.w[vid] + g) * dt; //dt velocity

        ps.p[vid] = x_t[vid] + k[i] * 0.5f;
    }
    else if(step == 3)
    {
        k[i] = (ps.v[vid] + l[i-1]) * dt; // position
        l[i] = (ps.f[vid] * ps.w[vid] + g) * dt; // velocity

        const Vector3 dt_p = (1.f/6.f) * (k[vid * 4] + 2.f * k[vid*4+1] + 2.f * k[vid*4+2] + k[vid*4+3]);
        const Vector3 dt_v = (1.f/6.f) * (l[vid * 4] + 2.f * l[vid*4+1] + 2.f * l[vid*4+2] + l[vid*4+3]);
        if(glm::length(dt_v) > 10) ps.mask[i] = 2;
        if(ps.mask[i] == 2) {
            ps.v[vid] = ps.v[vid] + g * dt;
            ps.p[vid] = x_t[vid] + ps.v[vid] * dt;
        }
        else {
            ps.v[vid] = ps.v[vid] + dt_v;
            ps.p[vid] = x_t[vid] + dt_p;
        }

    }
    ps.f[vid] *= 0.f;
}

__global__ void kenerl_semi_exicit_integration2(const int n, const scalar dt, const Vector3 g, GPU_ParticleSystem_Parameters ps, const scalar* w_max) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || ps.mask[i] == 0) return;
    Vector3 dt_v = ps.f[i] * ps.w[i] * dt;
    scalar dt_limit = 0.75f / sqrtf(w_max[i]);
    if(dt < dt_limit || ps.mask[i] == 2) // 0.75 is supposed to be equal to 2, its a magic number
    {
        ps.mask[i] = 2;
        dt_v = Vector3(0);
    }
    ps.v[i] += g * dt + dt_v;
    ps.p[i] += ps.v[i] * dt;
    ps.f[i] *= 0;
}

__global__ void kernel_reset_mask(const int n, GPU_ParticleSystem_Parameters ps)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || ps.mask[i] == 0 || ps.mask[i] == 3) return;
    ps.mask[i] = 1;
}

__global__ void kenerl_semi_exicit_integration3(const int n, const scalar dt, const Vector3 g, GPU_ParticleSystem_Parameters ps, const Vector3* last_v) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || ps.mask[i] == 0) return;
    const Vector3 dt_v = ps.f[i] * ps.w[i] * dt;
    if(glm::length(dt_v) > 10 && ps.mask[i] != 3) {
        ps.mask[i] = 2;
        ps.v[i] = last_v[i];
    }

    if(ps.mask[i] == 1) {
        ps.v[i] += g * dt + dt_v;
    }

    ps.p[i] += ps.v[i] * dt;
    ps.f[i] *= 0;
}

__global__ void kernel_inertia(const scalar dt, const Vector3 g, GPU_ParticleSystem_Parameters ps, Vector3* y, Vector3* last_v) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ps.nb_particles) return;
    ps.last_p[i] = ps.p[i]; // x^t-1 = x^t
    const Vector3 a_ext = g + ps.f[i] * ps.w[i];
    const Vector3 dt_v = ps.v[i] + a_ext * dt;
    last_v[i] = dt_v;
    y[i] = ps.p[i] + dt_v * dt;
}

void GPU_Mixed_VBD::step(const scalar dt) {
    const int n = nb_particles();
    const GPU_ParticleSystem_Parameters ps_param = get_parameters();
    kernel_reset_mask<<<(n+31) / 32, 32>>>(n, ps_param);

    // Compute inertia and save last_p (doesn't change v or p nor f)
    kernel_inertia<<<(n + 31)/32, 32>>>(dt,Dynamic::gravity(), ps_param,y->buffer, last_v->buffer);

    const scalar dt_exp = dt / static_cast<scalar>(explicit_it);
    for(int i = 0; i < explicit_it; ++i)
    {
        // eval forces
        // integrations Euler semi-implicit
        for(const GPU_Mixed_VBD_FEM* fem : _fems) fem->explicit_step(this, w_max, dt_exp);
        kenerl_semi_exicit_integration3<<<(n+31) / 32, 32>>>(n, dt_exp, Dynamic::gravity(), get_parameters(), last_v->buffer);/**/
        //kenerl_semi_exicit_integration2<<<(n+31) / 32, 32>>>(n, dt_exp, Dynamic::gravity(), get_parameters(), w_max->buffer);/**/

        //integration Runge-Kutta 4
        /*for(int j = 0; j < 4; ++j) {
            for(const GPU_Mixed_VBD_FEM* fem : _fems)
                fem->explicit_step(this, w_max, dt_exp);
            kernel_rk4<<<(n+31) / 32, 32>>>(n, dt_exp, Dynamic::gravity(), j, ps_param, l->buffer, k->buffer, rk4_last_p->buffer);
        }/**/
    }


    for(int j = 0; j < iteration; ++j) {
        // solve
        for(GPU_Dynamic* dynamic : _dynamics)
            dynamic->step(this, dt);

        for(GPU_Dynamic * constraint : _constraints)
            constraint->step(this, dt);
    }

    // velocity update
    kernel_velocity_update<<<(n + 31)/32, 32>>>(dt,ps_param);

}