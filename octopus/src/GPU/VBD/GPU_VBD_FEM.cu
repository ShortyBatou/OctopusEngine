#include "hip/hip_runtime.h"
#include "GPU/CUMatrix.h"
#include <glm/detail/func_matrix_simd.inl>
#include <Manager/Debug.h>
#include <Manager/Dynamic.h>
#include "GPU/VBD/GPU_VBD_FEM.h"

#include <random>
#include <set>
#include <GPU/GPU_FEM_Material.h>
#include <GPU/GPU_ParticleSystem.h>




__device__ void compute_f_H(
    const int n, const int r_vid,
    const Material_Data& mt, const GPU_ParticleSystem_Parameters ps, const int* topo,
    const Matrix3x3 &JX_inv, const scalar V, const Vector3* dN,
    Vector3& fi, Matrix3x3& H) {
    Matrix3x3 Jx(0.f);
    Matrix3x3 d2W_dF2[6];

    for (int i = 0; i < n; ++i) {
        Jx += glm::outerProduct(ps.p[topo[i]], dN[i]);
    }

    const Matrix3x3 F = Jx * JX_inv;
    const Vector3 dF_dx = glm::transpose(JX_inv) * dN[r_vid];

    // Compute force at vertex i
    const Matrix3x3 P = eval_pk1_stress(mt.material, mt.lambda, mt.mu, F);
    fi -= P * dF_dx * V;

    // Compute hessian
    eval_hessian(mt.material, mt.lambda, mt.mu, F, d2W_dF2);
    H += assemble_sub_hessian(dF_dx, V, d2W_dF2);
}

__device__ void vec_reduction(const int tid, const int block_size, const int v_size, scalar* s_data) {
    __syncthreads();
    int i,b;
    for(i=block_size/2, b=(block_size+1)/2; i > 0; b=(b+1)/2, i/=2) {
        if(tid < i) {
            for(int j = 0; j < v_size; ++j) {
                s_data[tid*v_size+j] += s_data[(tid+b)*v_size+j];
            }
            __syncthreads();
        }
        i = (b>i) ? b : i;
    }
}

__device__ void store_f_H_in_shared(const int tid, const Vector3& fi, const Matrix3x3& H, scalar* s_data) {
    int k = 0;
    for(int j = 0; j < 3; ++j) {
        s_data[tid * 12 + j] = fi[j];
        for(int i = 0; i < 3; ++i) {
            s_data[tid * 12 + 3 + k] = H[i][j];
            ++k;
        }
    }
}

__device__ void store_f_H_in_shared_sym(const int tid, const Vector3& fi, const Matrix3x3& H, scalar* s_data) {
    int k = 0;
    for(int j = 0; j < 3; ++j) {
        s_data[tid * 9 + j] = fi[j];
        for(int i = j; i < 3; ++i) {
            s_data[tid * 9 + 3 + k] = H[i][j];
            ++k;
        }
    }
}


__device__ Vector3 compute_correction(const int vid, const scalar damping, const scalar dt,
        GPU_ParticleSystem_Parameters ps, const Vector3* y,
        Vector3& fi, Matrix3x3& H) {
    // damping (velocity)
    fi -= damping / dt * H * (ps.p[vid] - ps.last_p[vid]);
    H  += damping / dt * H;

    // inertia (acceleration)
    const scalar mh2 = ps.m[vid] / (dt*dt);
    fi -= mh2 * (ps.p[vid] - y[vid]);
    H[0][0] += mh2; H[1][1] += mh2; H[2][2] += mh2;
    //scalar detH = glm::determinant(s_H);
    const scalar detH = abs(glm::determinant(H));
    return detH > 1e-6f ? glm::inverse(H) * fi : Vector3(0.f);
}

__global__ void kernel_vbd_solve_v1(
    const int n,
    const scalar damping,
    const scalar dt,
    const int offset,
    const Vector3* y,
    Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    if (blockIdx.x * blockDim.x + threadIdx.x >= n) return;

    // the group size depends on the number of element that contains these vertices
    // and the number of needed quadratures
    const int cid = offset + blockIdx.x; // vertex position in coloration
    const int block_size = owners.nb[cid];
    const int tid = threadIdx.x;
    if (tid >= block_size) return;

    const int e_off = owners.offset[cid] + tid; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];

    if(ps.mask[vid] == 0) return;

    Vector3 fi(0.f);
    Matrix3x3 H(0.f);
    for(int q = 0; q < fem.nb_quadrature; ++q) {
        const int qe_off = eid * fem.nb_quadrature + q;
        const int qv_off = q * fem.elem_nb_vert;
        compute_f_H(fem.elem_nb_vert, r_vid,
                    mt, ps, topo, fem.JX_inv[qe_off], fem.V[qe_off], fem.dN + qv_off,
                    fi, H);
    }

    // shared variable : f, H
    extern __shared__ scalar s_f_H[]; // size = block_size * 12 * sizeof(float)
    store_f_H_in_shared(tid, fi, H, s_f_H); // store f and h in shared memory
    vec_reduction(tid, block_size, 12, s_f_H); // reduction of fi Hi

    if (threadIdx.x == 0) {
        fi.x = s_f_H[0]; fi.y = s_f_H[1]; fi.z = s_f_H[2];
        H[0][0] = s_f_H[3]; H[0][1] = s_f_H[6]; H[0][2] = s_f_H[9 ];
        H[1][0] = s_f_H[4]; H[1][1] = s_f_H[7]; H[1][2] = s_f_H[10];
        H[2][0] = s_f_H[5]; H[2][1] = s_f_H[8]; H[2][2] = s_f_H[11];

        ps.p[vid] += compute_correction(vid, damping, dt, ps, y, fi, H);
    }
}

__global__ void kernel_vbd_solve_v2(
    const int n,
    const scalar damping,
    const scalar dt,
    const int offset,
    const Vector3* y,
    Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains these vertices
    // and the number of needed quadratures
    const int cid = offset + blockIdx.x; // vertex position in coloration
    const int block_size = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= block_size) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    if(ps.mask[vid] == 0) return;

    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Vector3 fi(0.f);
    Matrix3x3 H(0.f);

    compute_f_H(fem.elem_nb_vert, r_vid,
                    mt, ps, topo, fem.JX_inv[qe_off], fem.V[qe_off], fem.dN + qv_off,
                    fi, H);

    // shared variable : f, H
    extern __shared__ scalar s_f_H[]; // size = block_size * 12 * sizeof(float)
    store_f_H_in_shared(tid, fi, H, s_f_H); // store f and h in shared memory
    vec_reduction(tid, block_size, 12, s_f_H); // reduction of fi Hi

    if (threadIdx.x == 0) {
        fi.x = s_f_H[0]; fi.y = s_f_H[1]; fi.z = s_f_H[2];
        H[0][0] = s_f_H[3]; H[0][1] = s_f_H[6]; H[0][2] = s_f_H[9 ];
        H[1][0] = s_f_H[4]; H[1][1] = s_f_H[7]; H[1][2] = s_f_H[10];
        H[2][0] = s_f_H[5]; H[2][1] = s_f_H[8]; H[2][2] = s_f_H[11];

        ps.p[vid] += compute_correction(vid, damping, dt, ps, y, fi, H);
    }
}

__global__ void kernel_vbd_solve_v3(
    const int n,
    const scalar damping,
    const scalar dt,
    const int offset,
    const Vector3* y,
    Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains these vertices
    // and the number of needed quadratures
    const int cid = offset + blockIdx.x; // vertex position in coloration
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    if(ps.mask[vid] == 0) return;

    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;

    Vector3 fi(0.f);
    Matrix3x3 H(0.f);
    compute_f_H(fem.elem_nb_vert, r_vid,
                mt, ps, topo, fem.JX_inv[qe_off], fem.V[qe_off], fem.dN + qv_off,
                fi, H);


    // shared variable : f, H
    extern __shared__ scalar s_f_H[]; // size = block_size * 9 * sizeof(float)
    store_f_H_in_shared_sym(tid, fi, H, s_f_H);
    vec_reduction(tid, size_of_block, 9, s_f_H);

    if (threadIdx.x == 0) {
        fi.x = s_f_H[0]; fi.y = s_f_H[1]; fi.z = s_f_H[2];
        H[0][0] = s_f_H[3];
        H[1][0] = s_f_H[4]; H[1][1] = s_f_H[6];
        H[2][0] = s_f_H[5]; H[2][1] = s_f_H[7];  H[2][2] = s_f_H[8];
        // symmetry
        H[0][1] = H[1][0]; H[1][2] = H[2][1]; H[0][2] = H[2][0];

        ps.p[vid] += compute_correction(vid, damping, dt, ps, y, fi, H);
    }
}




__global__ void kernel_vbd_compute_residual(
    const int n,
    const scalar damping,
    const scalar dt,
    const int offset,
    const Vector3* y,
    Vector3* r,
    Material_Data mt,
    GPU_ParticleSystem_Parameters ps,
    GPU_FEM_Pameters fem,
    GPU_Owners_Parameters owners
) {
    // global id
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= n) return;

    // the group size depends on the number of element that contains these vertices
    // and the number of needed quadratures
    const int cid = offset + blockIdx.x; // vertex position in coloration
    const int size_of_block = owners.nb[cid] * fem.nb_quadrature;
    const int tid = threadIdx.x; // thread id in block
    if (tid >= size_of_block) return;

    const int qid = tid % fem.nb_quadrature; // quadrature number
    const int e_off = owners.offset[cid] + tid / fem.nb_quadrature; // offset in buffer to find the right element
    const int eid = owners.eids[e_off]; // element id
    const int r_vid = owners.ref_vid[e_off]; // vertex id in this element

    const int *topo = fem.topology + eid * fem.elem_nb_vert; // offset the pointer at the start of the element's topology
    const int vid = topo[r_vid];
    if(ps.mask[vid] == 0) return;

    const int qe_off = eid * fem.nb_quadrature + qid;
    const int qv_off = qid * fem.elem_nb_vert;
    Vector3 fi(0.f);
    Matrix3x3 H(0.f);
    compute_f_H(fem.elem_nb_vert, r_vid,
                mt, ps, topo, fem.JX_inv[qe_off], fem.V[qe_off], fem.dN + qv_off,
                fi, H);

    // shared variable : f, H
    extern __shared__ scalar s_f_H[]; // size = block_size * 12 * sizeof(float)
    store_f_H_in_shared_sym(tid, fi, H, s_f_H);
    vec_reduction(tid, size_of_block, 9, s_f_H);

    if (threadIdx.x == 0) {
        fi = Vector3(0);
        fi.x = s_f_H[0]; fi.y = s_f_H[1]; fi.z = s_f_H[2];
        H[0][0] = s_f_H[3];
        H[1][0] = s_f_H[4]; H[1][1] = s_f_H[6];
        H[2][0] = s_f_H[5]; H[2][1] = s_f_H[7];  H[2][2] = s_f_H[8];
        // symmetry
        H[0][1] = H[1][0]; H[1][2] = H[2][1]; H[0][2] = H[2][0];

        // damping (velocity)
        fi -= damping / dt * H * (ps.p[vid] - ps.last_p[vid]);
        H  += damping / dt * H;

        // intertia (accellearation)
        const scalar mh2 = ps.m[vid] / (dt*dt);
        fi -= mh2 * (ps.p[vid] - y[vid]);
        H  += mh2 * Matrix3x3(1.);
        r[vid] = fi;
    }
}

std::vector<Vector3> GPU_VBD_FEM::get_forces(const GPU_ParticleSystem *ps, const scalar dt) const {
    for(int c = 0; c < d_thread->nb_kernel; ++c) {
        kernel_vbd_compute_residual<<<d_thread->grid_size[c], d_thread->block_size[c]>>>(
            d_thread->nb_threads[c], damping, dt, d_thread->offsets[c],
             y->buffer, r->buffer, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters()
        );
    }
    std::vector<Vector3> residual(ps->nb_particles());
    r->get_data(residual);
    return residual;
}


GPU_VBD_FEM::GPU_VBD_FEM(const Element &element, const Mesh::Topology &topology, const Mesh::Geometry &geometry,
                         const Material& material, const scalar &young, const scalar &poisson, const scalar& damping,
                         const VBD_Version& v) :
    GPU_FEM(element, geometry, topology, young, poisson, material), version(v), y(nullptr)
{
    const int nb_vertices = static_cast<int>(geometry.size());
    this->damping = damping;
    d_owners = new GPU_Owners_Data();
    r = new Cuda_Buffer(nb_vertices, Vector3(0.f));
    std::vector<std::vector<int>> e_owners;
    std::vector<std::vector<int>> e_ref_id;
    build_graph_color(topology, nb_vertices, _colors,e_owners,e_ref_id);
    sort_by_color(nb_vertices, _colors, e_owners, e_ref_id);
    shared_sizes.push_back(10);
}

void GPU_VBD_FEM::sort_by_color(const int nb_vertices, const std::vector<int>& colors, const std::vector<std::vector<int>>& e_owners, const std::vector<std::vector<int>>& e_ref_id)
{
    // sort by color
    std::vector<int> ref_id;
    std::vector<int> owners;
    std::vector<int> nb_owners;
    std::vector<int> owners_offset;
    // sort neighbors
    for(int c = 0; c < d_thread->nb_kernel; ++c) {
        int n_max = 1;
        int nb_vert = 0;
        int n = static_cast<int>(nb_owners.size());
        for(int i = 0; i < nb_vertices; ++i) {
            if(c != colors[i]) continue;
            owners_offset.push_back(static_cast<int>(owners.size()));
            owners.insert(owners.end(), e_owners[i].begin(), e_owners[i].end());
            ref_id.insert(ref_id.end(), e_ref_id[i].begin(), e_ref_id[i].end());
            nb_owners.push_back(static_cast<int>(e_owners[i].size()));

            n_max = std::max(n_max, nb_owners.back());
            nb_vert ++;
        }
        d_thread->grid_size.push_back(nb_vert);
        d_thread->block_size.push_back(n_max);
        shared_sizes.push_back(n_max * sizeof(scalar));
        d_thread->nb_threads.push_back(nb_vert * n_max);
        d_thread->offsets.push_back(n);
    }
    d_owners->cb_nb = new Cuda_Buffer(nb_owners);
    d_owners->cb_eids = new Cuda_Buffer(owners);
    d_owners->cb_offset = new Cuda_Buffer(owners_offset);
    d_owners->cb_ref_vid = new Cuda_Buffer(ref_id);
}


void GPU_VBD_FEM::build_graph_color(const Mesh::Topology &topology, const int nb_vertices,
    std::vector<int> &colors, std::vector<std::vector<int>>& e_neighbors, std::vector<std::vector<int>>& e_ref_id) const
{
    std::vector<std::set<int> > neighbors(nb_vertices);
    e_neighbors.resize(nb_vertices);
    e_ref_id.resize(nb_vertices);
    // for each vertice get all its neighboors
    for (int i = 0; i < topology.size(); i += d_fem->elem_nb_vert) {
        int eid = i / d_fem->elem_nb_vert;
        for (int j = 0; j < d_fem->elem_nb_vert; ++j) {
            e_neighbors[topology[i + j]].push_back(eid);
            e_ref_id[topology[i+j]].push_back(j);
            // all vertices inside an element are linked
            for (int k = 0; k < d_fem->elem_nb_vert; ++k) {
                if (k == j) continue;
                neighbors[topology[i + j]].insert(topology[i + k]);
            }
        }
    }

    int max_neighbors = 0;
    d_thread->nb_kernel = 1;
    colors.resize(nb_vertices, -1);
    std::vector<int> available(64, true);
    for (int i = 0; i < nb_vertices; ++i) {
        // for all vertices, check the neighbor elements colors
        max_neighbors = std::max(static_cast<int>(e_neighbors[i].size()), max_neighbors);
        for (const int n: neighbors[i]) {
            if (colors[n] != -1) available[colors[n]] = false;
        }
        for (int c = 0; c < available.size(); ++c) {
            if (available[c]) {
                d_thread->nb_kernel = std::max(d_thread->nb_kernel, c);
                colors[i] = c;
                break;
            }
        }
        std::fill(available.begin(), available.end(), true);
    }
    d_thread->nb_kernel++;
    std::cout << "NB color: " << d_thread->nb_kernel << "  NB neighbors : " << max_neighbors << std::endl;
}

void GPU_VBD_FEM::step(GPU_ParticleSystem* ps, const scalar dt) {
    std::vector<int> kernels(d_thread->nb_kernel);
    std::iota(kernels.begin(), kernels.end(), 0);
    std::shuffle(kernels.begin(), kernels.end(), std::mt19937());
    unsigned int s;
    for(const int c : kernels) {
        switch(version) {
            case Base :
                s = d_thread->block_size[c] * 12 * sizeof(scalar);
                kernel_vbd_solve_v1<<<d_thread->grid_size[c], d_thread->block_size[c], s>>>(
                    d_thread->nb_threads[c], damping, dt, d_thread->offsets[c],
                      y->buffer, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters());
            break;
            case Threaded_Quadrature :
                s = d_thread->block_size[c] * d_fem->nb_quadrature * 12 * sizeof(scalar);
                kernel_vbd_solve_v2<<<d_thread->grid_size[c], d_thread->block_size[c] * d_fem->nb_quadrature, s>>>(
                    d_thread->nb_threads[c] * d_fem->nb_quadrature, damping, dt, d_thread->offsets[c],
                     y->buffer, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters()
                );
            break;
            case Reduction_Symmetry :
                s = d_thread->block_size[c] * d_fem->nb_quadrature * 9 * sizeof(scalar);
                kernel_vbd_solve_v3<<<d_thread->grid_size[c], d_thread->block_size[c] * d_fem->nb_quadrature, s>>>(
                    d_thread->nb_threads[c] * d_fem->nb_quadrature, damping, dt, d_thread->offsets[c],
                    y->buffer, *d_material, ps->get_parameters(), get_fem_parameters(), get_owners_parameters()
                );
            break;
        }
    }
}